
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <time.h>

__global__ void op_single(int n, double r1) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	double total = 0.0;
	for (int i = index; i < n; i += step) {
		total += atan(r1);
	}
	printf("tot single: %lf\n", total);
}

__global__ void op_multi(int n, double r1, float *device_arr) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < n; i += step) {
		device_arr[i] += atan(r1);
	}
}

int main(int argc, char *argv[]) {
	srand(42);
	double r1 = ((double) rand()) / ((double) (RAND_MAX));
	int n_iterations = 4194304;

	// multi thread
		printf("running multi\n");
		int block_size = 1024;

		int num_blocks = n_iterations / block_size;
		int n_threads = num_blocks * block_size;
		int blocks_per_grid = n_iterations / block_size;

		clock_t t = clock();
		size_t mem_size = n_iterations * sizeof(float);
		printf("using byte: %d\n", (int) mem_size);
		float *host_arr = (float *) malloc(mem_size);
		for (int i = 0; i < n_iterations; i++) {
			host_arr[i] = 0.0;
		}

		float *device_arr = NULL;
		hipMalloc((void **) &device_arr, mem_size);
		hipMemcpy(device_arr, host_arr, mem_size, hipMemcpyHostToDevice);

		clock_t t2 = clock();
		printf("num grids: %d, num threads: %d\n", blocks_per_grid, block_size);
		op_multi<<<blocks_per_grid, block_size>>>(n_iterations, r1, device_arr);
		t2 = clock() - t2;

		hipMemcpy(host_arr, device_arr, mem_size, hipMemcpyDeviceToHost);
		float sum = 0.0;
		for (int i = 0; i < n_iterations; i++) {
			sum += host_arr[i];
		}
		t = clock() - t;
		printf("tot multi: %f\n", sum);

		hipFree(device_arr);
		free(host_arr);
		printf("It took GPU multi with malloc: %f s.\n", (((float) t) / 1000000));
		printf("It took GPU multi kernel only: %f s.\n", (((float) t2) / 1000000));

		//single thread
		printf("running single\n");
		 t = clock();
		op_single<<<1, 1>>>(n_iterations, r1);
		hipDeviceSynchronize();
		printf("It took GPU single %f s.\n", (((float) clock() - t) / 1000000));
	return 0;
}
